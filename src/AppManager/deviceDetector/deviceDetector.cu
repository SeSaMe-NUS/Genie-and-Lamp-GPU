/* *
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>

/**
 * print error text
 */
static void HandleError(hipError_t err, const char *file, int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		exit(EXIT_FAILURE);
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
void IDDetector(void) {

	hipDeviceProp_t prop;
	int dev;

	HANDLE_ERROR(hipGetDevice(&dev));
	printf("ID of current CUDA device:  %d\n", dev);

	memset(&prop, 0, sizeof(hipDeviceProp_t));
	prop.major = 1;
	prop.minor = 3;
	HANDLE_ERROR(hipChooseDevice(&dev, &prop));
	printf("ID of CUDA device closest to revision 1.3:  %d\n", dev);

	HANDLE_ERROR(hipSetDevice(dev));

}

void parameterDetector(void) {

	hipDeviceProp_t prop;
	int count;
	HANDLE_ERROR(hipGetDeviceCount(&count));
	for (int i = 0; i < count; i++) {
		HANDLE_ERROR(hipGetDeviceProperties(&prop, i));
		printf("   --- General Information for device %d -\n", i);
		printf("Name:  %s\n", prop.name);
		printf("Compute capability:  %d.%d\n", prop.major, prop.minor);
		printf("Clock rate:  %d\n", prop.clockRate);
		printf("Device copy overlap:  ");
		if (prop.deviceOverlap)
			printf("Enabled\n");
		else
			printf("Disabled\n");
		printf("Kernel execution timeout :  ");
		if (prop.kernelExecTimeoutEnabled)
			printf("Enabled\n");
		else
			printf("Disabled\n");

		printf("   --- Memory Information for device %d ---\n", i);
		printf("Total global mem:  %ld\n", prop.totalGlobalMem);
		printf("Total constant Mem:  %ld\n", prop.totalConstMem);
		printf("Max mem pitch:  %ld\n", prop.memPitch);
		printf("Texture Alignment:  %ld\n", prop.textureAlignment);

		printf("   - MP Information for device %d -\n", i);
		printf("Multiprocessor count:  %d\n", prop.multiProcessorCount);
		printf("Shared mem per mp:  %ld\n", prop.sharedMemPerBlock);
		printf("Registers per mp:  %d\n", prop.regsPerBlock);
		printf("Threads in warp:  %d\n", prop.warpSize);
		printf("Max threads per block:  %d\n", prop.maxThreadsPerBlock);
		printf("Max thread dimensions:  (%d, %d, %d)\n", prop.maxThreadsDim[0],
				prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("Max grid dimensions:  (%d, %d, %d)\n", prop.maxGridSize[0],
				prop.maxGridSize[1], prop.maxGridSize[2]);
		printf("\n");
	}
}

void runDevicDetector() {
	IDDetector();
	parameterDetector();
}
