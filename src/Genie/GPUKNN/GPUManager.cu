#include "hip/hip_runtime.h"
/*
 * GPUManager.cpp
 *
 *  Created on: Apr 7, 2014
 *      Author: yuxin
 */

#include <iostream>
#include <fstream>
#include <sstream>
#include <assert.h>

#include <hip/hip_runtime_api.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/scan.h>

#include "GPUManager.h"
#include "generalization.h"

using namespace std;
using namespace thrust;

GPUManager::GPUManager() :
		invert_list_spec_host(0)
{

	init_parameters_default();

}

GPUManager::GPUManager(GPUSpecification& gpu_spec)
{
	conf_GPUManager_GPUSpecification(gpu_spec);
}

/**
 * GPUSpecification: specify the configuration for GPUManager, i.e. index file path etc.
 * rand_invert_list_size: if inverted index file is empty, generate the data inverted index by random
 */
GPUManager::GPUManager(GPUSpecification& query_spec, int rand_invert_list_size) :
		invert_list_spec_host(query_spec.totalDimension)
{
	srand(1);

	invert_list_spec_host.numOfDocToExpand = query_spec.numOfDocToExpand;

	if (invert_list_spec_host.numOfDocToExpand <= 0)
	{
		cout << "cannot handle numOfDocToExpand is equal or below 0" << endl;
		exit(1);
	}

	for (int i = 0; i < invert_list_spec_host.totalDimension; i++)
	{
		invert_list_spec_host.indexDimensionEntry[i] =
				query_spec.indexDimensionEntry[i];

	}

	this->set_query_DefaultDisFuncType(query_spec.default_disfuncType);
	this->set_query_DefaultDistBound(query_spec.default_upwardDistBound,
			query_spec.default_downwardDistBound);

	// read inverted list
	if (query_spec.invertedListPath.empty()
			&& query_spec.invertedIndexLengthFile.empty())
	{
		invert_list_spec_host.maxFeatureID = rand_invert_list_size;
		rand_inv_list();
	}
	else
	{
		// read inverted list from file
		//this->get_invert_list_from_file(query_spec.invertedListPath,query_spec.invertedIndexLengthFile);
		int numofDim = 0;		//out result
		int maxValuePerDim = 0;		//output result
		int maxFeatureID = 0;
		get_invert_list_from_binary_file(query_spec.invertedListPath, numofDim,
				maxValuePerDim, maxFeatureID);//Note: also set the maxFeatureID

		this->max_number_of_dimensions = numofDim;
		this->max_value_per_dimension = maxValuePerDim;
		this->invert_list_spec_host.maxFeatureID = maxFeatureID;
	}

	clear_query_memory();

	exec_time.resize(15, 0);
}

GPUManager::~GPUManager()
{

	//freeQueryInfoDevice(d_query_info);
	clear_query_memory();

}

void GPUManager::init_parameters_default()
{
	this->num_of_rounds = 0;

	this->max_number_of_dimensions = 0;
	this->max_value_per_dimension = 0;
	this->invert_list_spec_host.maxFeatureID = 0;
	this->query_initialized = false;
	sumQueryDims = 0;
	default_disfuncType = 2;
	default_upwardDistBound = 0;
	default_downwardDistBound = 0;
	topK = 0;
}

void GPUManager::conf_GPUManager_GPUSpecification(GPUSpecification& gpu_spec)
{

	invert_list_spec_host.init_InvertListSpecGPU(gpu_spec.totalDimension);
	invert_list_spec_host.numOfDocToExpand = gpu_spec.numOfDocToExpand;

	if (invert_list_spec_host.numOfDocToExpand <= 0)
	{
		cout << "cannot handle numOfDocToExpand is equal or below 0" << endl;
		exit(1);
	}

	for (int i = 0; i < invert_list_spec_host.totalDimension; i++)
	{
		invert_list_spec_host.indexDimensionEntry[i] =
				gpu_spec.indexDimensionEntry[i];

	}

	this->set_query_DefaultDisFuncType(gpu_spec.default_disfuncType);
	this->set_query_DefaultDistBound(gpu_spec.default_upwardDistBound,
			gpu_spec.default_downwardDistBound);

	// read inverted list from file
	int numofDim = 0;		//out result
	int maxValuePerDim = 0;		//output result
	int maxFeatureID = 0;

	//this function also set the invert_list_spec_host.maxFeatureID which determinese the size of count&ACD table
	get_invert_list_from_binary_file(gpu_spec.invertedListPath, numofDim,
			maxValuePerDim, maxFeatureID);

	this->max_number_of_dimensions = numofDim;
	this->max_value_per_dimension = maxValuePerDim;
	this->invert_list_spec_host.maxFeatureID = maxFeatureID;

	clear_query_memory();

	exec_time.resize(15, 0);
}

void GPUManager::clear_query_memory()
{

	free_queryInfo_vec_onDevice(d_query_info);
	d_query_info.clear();
	d_query_info.shrink_to_fit();

	for (int i = 0; i < this->h_query_info.size(); i++)
	{
		delete h_query_info[i];
	}
	h_query_info.clear();

	sumQueryDims = 0;

	d_query_feature.clear();
	d_query_feature.shrink_to_fit();

	d_valid_query.clear();
	d_valid_query.shrink_to_fit();
	this->query_initialized = false;

}

/**
 *
 * TODO:
 *
 *
 */
void GPUManager::init_GPU_query(vector<GpuQuery>& query_set)
{
	clear_query_memory();

	if (query_set.empty())
	{
		exit(1);
	}
	else
	{
		this->invert_list_spec_host.numOfQuery = query_set.size();
		//host_vector<QueryInfo*> h_query_info;

		h_query_info.reserve(this->invert_list_spec_host.numOfQuery);
		hdmap.timer = 0;
		for (int i = 0; i < this->invert_list_spec_host.numOfQuery; i++)
		{
			QueryInfo *queryInfo = new QueryInfo(query_set[i]);
			sumQueryDims += queryInfo->numOfDimensionToSearch;

			// TODO: init keyword_indexMapping YIWEI;
			//USE Mapping function

			for (int j = 0; j < queryInfo->numOfDimensionToSearch; j++)
			{
				int dim = queryInfo->searchDim[j];
				int key = (int) queryInfo->keyword[j] + 0.5
					+ dim * max_value_per_dimension;
				try
				{
					queryInfo->keyword_indexMapping[j] = hdmap.get_index(key);
				} catch (int code)
				{
					queryInfo->keyword_indexMapping[j] = hdmap.get_index(
							hdmap.get_fuzzy_lower_key(key));
				}
			}
			h_query_info.push_back(queryInfo);
		}
		printf("HDMap mapping time, HDMap.timer: %f\n", hdmap.timer);

		d_query_info.reserve(h_query_info.size());

		// copy queryInfo to gpu
		copy_QueryInfo_vec_fromHostToDevice(h_query_info, d_query_info);

	}
	//here!: test wether correctly initialized
	initialize_dev_constMem(invert_list_spec_host);
	printf("#total number of queries in GPUMananger %d\n",
			invert_list_spec_host.numOfQuery);

	d_indexDimensionEntry_vec.clear();
	d_indexDimensionEntry_vec.reserve(invert_list_spec_host.totalDimension);

	for (int i = 0; i < invert_list_spec_host.totalDimension; i++)
	{
		d_indexDimensionEntry_vec.push_back(
				invert_list_spec_host.indexDimensionEntry[i]);
	}

	// checking memory size
	if ((double) invert_list_spec_host.maxFeatureID
			* invert_list_spec_host.numOfQuery * sizeof(QueryFeatureEnt)
			> std::numeric_limits<int>::max())
	{
		cout << "too much memory is used for query feature mapping" << endl;
		exit(1);
	}

	//d_query_feature = new device_vector<QueryFeatureEnt>( (unsigned int) invert_list_spec_host.maxFeatureID * invert_list_spec_host.numOfQuery );
	d_query_feature.clear();
	d_query_feature.resize(
			(unsigned int) invert_list_spec_host.maxFeatureID
					* invert_list_spec_host.numOfQuery);
	d_query_feature.shrink_to_fit();
	//d_valid_query = new device_vector<int>( invert_list_spec_host.numOfQuery, 1 );
	d_valid_query.clear();
	d_valid_query.resize(invert_list_spec_host.numOfQuery, 1);
	d_valid_query.shrink_to_fit();
	num_of_rounds = 0;

	this->query_initialized = true;
	cout << "finish initialization" << endl;
}

struct gzero
{
	__host__ __device__
	bool operator()(const QueryFeatureEnt &x)
	{
		return x.count > 0;
	}
};

bool GPUManager::bi_direction_query_KernelPerDim(int threshold, int topKValue,
		vector<Result>& _result)
{

	check_query_parameter();

	hipProfilerStart();
	hipEvent_t start, stop;
	hipEvent_t gs, ge;
	float elapsedTime;
	float elapseTimeForCount;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventCreate(&gs);
	hipEventCreate(&ge);

	//cout	<< "GPUManager::bi_direction_query(): 0 -- compute_mapping_saving_pos_KernelPerDim()"
	//		<< endl;
	hipEventRecord(gs, 0);
	hipEventRecord(start, 0);

	compute_mapping_saving_pos_KernelPerDim_template<<<sumQueryDims,
			THREAD_PER_BLK>>>(raw_pointer_cast(this->d_query_info.data()),
			raw_pointer_cast(this->d_invert_list.data()),
			raw_pointer_cast(this->hdmap.raw_value().data()), //raw_pointer_cast(this->d_invert_list_idx.data()),
			raw_pointer_cast(this->d_query_feature.data()),	//QueryFeatureEnt* query_feature, i.e. count&ACD table
			false, max_value_per_dimension,
			raw_pointer_cast(this->d_indexDimensionEntry_vec.data()),
			DataToIndex_keywordMap_bucketUnit(),
			IndexToData_lastPosMap_bucketUnit(), Lp_distance());

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	exec_time[0] += elapsedTime;

	//cout	<< "GPUManager::bi_direction_query(): 1 -- prefix_count_KernelPerQuery()"
	//		<< endl;
	device_vector<int> threshold_count(
			invert_list_spec_host.numOfQuery * THREAD_PER_BLK, 0);// prefix count for each thread
	device_vector<int> query_result_count(invert_list_spec_host.numOfQuery, 0);	// prefix count for each block (i.e. each query)
	elapseTimeForCount = 0;

	hipEventRecord(start, 0);
	prefix_count_KernelPerQuery<<<this->invert_list_spec_host.numOfQuery,
			THREAD_PER_BLK, THREAD_PER_BLK * sizeof(int)>>>(
			raw_pointer_cast(this->d_query_feature.data()),
			raw_pointer_cast(threshold_count.data()),
			raw_pointer_cast(query_result_count.data()), threshold);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	elapseTimeForCount += elapsedTime;
	//exec_time[7] += elapsedTime;

	hipEventRecord(start, 0);
	thrust::inclusive_scan(threshold_count.begin(), threshold_count.end(),
			threshold_count.begin()); // per thread inclusive scan
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	elapseTimeForCount += elapsedTime;
	//	exec_time[8] += elapsedTime;
	hipEventRecord(start, 0);
	thrust::inclusive_scan(query_result_count.begin(), query_result_count.end(),
			query_result_count.begin()); // per block inclusive scan
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	elapseTimeForCount += elapsedTime;
	//exec_time[9] += elapsedTime;
	exec_time[1] += elapseTimeForCount;

	//cout	<< "GPUManager::bi_direction_query(): 2 -- output_result_bidrection_search_KernelPerQuery()"
	//		<< endl;
	device_vector<Result> d_result_ub_sorted(
			threshold_count[threshold_count.size() - 1]);
	//cout <<" the Count sum is:"<<query_result_count[invert_list_spec_host.numOfQuery-1]<<endl;
	hipEventRecord(start, 0);

	output_result_bidrection_search_KernelPerQuery_template<<<
			this->invert_list_spec_host.numOfQuery, THREAD_PER_BLK,
			3 * max_number_of_dimensions * sizeof(float)>>>(
			raw_pointer_cast(this->d_query_feature.data()),
			raw_pointer_cast(this->d_query_info.data()),
			raw_pointer_cast(threshold_count.data()),
			raw_pointer_cast(d_result_ub_sorted.data()), //record the lower bound and upper bound
			threshold, raw_pointer_cast(this->d_indexDimensionEntry_vec.data()),
			//raw_pointer_cast(this->maxDomainForAllDimension.data())//remove
			DataToIndex_keywordMap_bucketUnit(),
			IndexToData_lastPosMap_bucketUnit(), Lp_distance());

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	exec_time[2] += elapsedTime;

	//cout << "GPUManager::bi_direction_query(): 3 -- sort d_result_lb & ub"
	//		<< endl;
	//device_vector<Result> d_result_temp(d_result_ub_sorted.size());
	device_vector<Result> d_result_lb_sorted(d_result_ub_sorted.begin(),
			d_result_ub_sorted.end());
	hipEventRecord(start, 0);
	//thrust::sort(d_result_ub_sorted.begin(), d_result_ub_sorted.end(),
	//		Ubcomapre());
	//thrust::sort(d_result_lb_sorted.begin(), d_result_lb_sorted.end(),
	//		Lbcompare());
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	exec_time[3] += elapsedTime;

	//=====================Old impl=================================
//
//	//cout
//	//		<< "GPUManager::bi_direction_query(): 4 -- terminate_check_KernelPerQuery()"
//	//		<< endl;
//	device_vector<Result> d_result_temp(d_result_ub_sorted.size());//create temp buffer for checking
//	hipEventRecord(start, 0);
//
//
//	terminateCheck_kSelection_KernelPerQuery<<<this->invert_list_spec_host.numOfQuery,	THREAD_PER_BLK>>>(
//			raw_pointer_cast(d_result_ub_sorted.data()),
//			//raw_pointer_cast(d_result_lb_sorted.data()),
//			raw_pointer_cast(d_result_temp.data()),
//			raw_pointer_cast(query_result_count.data()),
//			raw_pointer_cast(d_valid_query.data()), topKValue
//			);
//	int terminate_sum = thrust::reduce(d_valid_query.begin(),
//			d_valid_query.end());
//
//
//	hipEventRecord(stop, 0);
//	hipEventSynchronize(stop);
//	hipEventElapsedTime(&elapsedTime, start, stop);
//	exec_time[4] += elapsedTime;
	//=====================Old impl=================================

	//=============================== Yiwei's Bucket Impl ========================
	//cout << "GPUManager::bi_direction_query(): 4 -- terminate_check_KernelPerQuery_Bucket()" << endl;
	int number_of_parts = this->invert_list_spec_host.numOfQuery;
	hipEventRecord(start, 0);
	terminateCheck_kSelection_KernelPerQuery_Bucket(&d_result_ub_sorted,
			&query_result_count, number_of_parts, topKValue,
			raw_pointer_cast(d_valid_query.data()));
	int terminate_sum = thrust::reduce(d_valid_query.begin(),
			d_valid_query.end());
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	exec_time[4] += elapsedTime;
	//=============================== Yiwei's Bucket Impl ========================

	//cout<< "GPUManager::bi_direction_query(): 4 -- extract_topK_KernelPerQuery()"<< endl;
	device_vector<Result> d_result(
			this->invert_list_spec_host.numOfQuery * topKValue);
	hipEventRecord(start, 0);
	extract_topK_KernelPerQuery<<<this->invert_list_spec_host.numOfQuery,
			THREAD_PER_BLK>>>(raw_pointer_cast(d_result_ub_sorted.data()),
			raw_pointer_cast(query_result_count.data()),
			raw_pointer_cast(d_result.data()), topKValue);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	exec_time[5] += elapsedTime;

	hipEventRecord(ge, 0);
	hipEventSynchronize(ge);
	hipEventElapsedTime(&elapsedTime, gs, ge);
	exec_time[6] += elapsedTime;

	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipProfilerStop();

	num_of_rounds++;

	_result.resize(d_result.size());
	HANDLE_ERROR(
			hipMemcpy(&_result[0], raw_pointer_cast(d_result.data()),
					d_result.size() * sizeof(Result), hipMemcpyDeviceToHost));

	// no more query should be made to this batch
	if (terminate_sum == 0)
	{
		this->query_initialized = false;
	}

	return terminate_sum == 0;
}

//==============================================================================
void GPUManager::point_query(vector<Result>& result, int threshold)
{

	check_query_parameter();

	compute_mapping_saving_pos_KernelPerDim_template<<<
			this->invert_list_spec_host.numOfQuery, THREAD_PER_BLK,
			max_number_of_dimensions>>>(
			raw_pointer_cast(this->d_query_info.data()),
			raw_pointer_cast(this->d_invert_list.data()),
			raw_pointer_cast(this->hdmap.raw_value().data()), //raw_pointer_cast(this->d_invert_list_idx.data()),
			raw_pointer_cast(this->d_query_feature.data()), true,
			max_value_per_dimension,
			raw_pointer_cast(this->d_indexDimensionEntry_vec.data()),
			DataToIndex_keywordMap_bucketUnit(),
			IndexToData_lastPosMap_bucketUnit(), Lp_distance());

	device_vector<int> threshold_count(
			invert_list_spec_host.numOfQuery * THREAD_PER_BLK, 0);// prefix count for each thread
	device_vector<int> query_result_count(invert_list_spec_host.numOfQuery, 0);	// prefix count for each block (query)

	prefix_count_KernelPerQuery<<<this->invert_list_spec_host.numOfQuery,
			THREAD_PER_BLK>>>(raw_pointer_cast(this->d_query_feature.data()),
			raw_pointer_cast(threshold_count.data()),
			raw_pointer_cast(query_result_count.data()), threshold);

	thrust::inclusive_scan(threshold_count.begin(), threshold_count.end(),
			threshold_count.begin()); // per thread inclusive scan
	thrust::inclusive_scan(query_result_count.begin(), query_result_count.end(),
			query_result_count.begin()); // per block inclusive scan

	device_vector<Result> d_result(threshold_count[threshold_count.size() - 1]);

	output_result_point_search<<<this->invert_list_spec_host.numOfQuery,
			THREAD_PER_BLK>>>(raw_pointer_cast(this->d_query_feature.data()),
			raw_pointer_cast(threshold_count.data()),
			raw_pointer_cast(d_result.data()), threshold);

	result.resize(d_result.size());
	HANDLE_ERROR(
			hipMemcpy(&result[0], raw_pointer_cast(d_result.data()),
					d_result.size() * sizeof(Result), hipMemcpyDeviceToHost));

	this->query_initialized = false; // make sure no other query is called
}

/**
 * retrieve invert list from binary given the filename, and read the number of dimension and max value per
 *  dimension from the binary file
 */
void GPUManager::get_invert_list_from_binary_file(string filename,
		int& _numberOfDim, int& _maxValuePerDim, int& _maxFeatureID)
{
	// initialize the previousKey to be 0;
	int previousKey = -1;
	int key;
	int numberOfFeatures;
	int keyPosition = 0;
	int offset = 0;
	_maxFeatureID = -1;
	hdmap.timer = 0;

	cout << "file name is: " << filename << endl;

	vector<InvlistEnt> featureIdList;
	vector<int> featureIdListIndex;

	clock_t start = clock();
	ifstream inFile(filename.c_str(), ios::in | ios::binary);

	if (!inFile)
	{
		cerr << "error : unable to open input file 1: " << inFile << endl;
	}
	else if (!inFile.is_open())
	{
		cerr << "error : unable to open input file 2: " << inFile << endl;
	}
	else
	{
		int numberOfDim;
		inFile.read((char*) (&numberOfDim), sizeof(int));
		_numberOfDim = numberOfDim;

		int maxValuePerDim;
		inFile.read((char*) (&maxValuePerDim), sizeof(int));
		_maxValuePerDim = maxValuePerDim;

		// assert( MAX_DIM_VALUE == maxValuePerDim );

		// read the key, if reach the end of the file, jump out
		while (inFile.read((char*) (&key), sizeof(int)))
		{
			int dim_value = key % maxValuePerDim;
			int dim = key / maxValuePerDim;

			if (dim >= invert_list_spec_host.totalDimension)
			{
				cerr << "error::dim is outside of maximum possible dimensions"
						<< endl;
				exit(1);
			}

			if (dim_value
					< invert_list_spec_host.indexDimensionEntry[dim].minDomain
					|| dim_value
							> invert_list_spec_host.indexDimensionEntry[dim].maxDomain)
			{
				cerr
						<< "error:input invert list has problem for dimension values, should be between min and max dimension value"
						<< endl;
				exit(1);
			}

			//	assert(dim<invert_list_spec_host.totalDimension &&("error::dim is outside of maximum possible dimensions\n"));

			/*if ( dim > invert_list_spec_host.totalDimension -1 )
			 {
			 cout << "dim: " << dim << endl;
			 cout << "key: " << key << endl;
			 cout << "read file problem" << endl;
			 }*/

			//assert((dim_value>=invert_list_spec_host.indexDimensionEntry[dim].minDomain &&
			//			dim_value<=invert_list_spec_host.indexDimensionEntry[dim].maxDomain)&&
			//		("error:input invert list has problem for dimension values, should be between min and max dimension value\n"));
			/*if ( dim_value < invert_list_spec_host.minDomainForAllDimension[dim] ||
			 dim_value > invert_list_spec_host.maxDomainForAllDimension[dim] )
			 {
			 cout << "input invert list has problem for dimension values, should be between min and max dimension value" << endl;
			 cout << "dimension " << dim << " min: " << invert_list_spec_host.minDomainForAllDimension[dim] << " and max: " << invert_list_spec_host.maxDomainForAllDimension[dim] << endl;
			 cout << "value is: " << dim_value << endl;
			 cout << "key: " << key << endl;
			 exit(1);
			 }*/

			// read the number of features
			inFile.read((char*) (&numberOfFeatures), sizeof(int));

			// read the feature ids, and fill in the featureIdList
			if (numberOfFeatures > 0)
			{
				int featureID;
				for (int j = 0; j < numberOfFeatures; j++)
				{

					// read the feature ID list
					inFile.read((char*) (&featureID), sizeof(int));
					InvlistEnt new_ent = featureID;
					if (new_ent > _maxFeatureID)
					{
						_maxFeatureID = new_ent;
					}
					featureIdList.push_back(new_ent);

				}

			}

			// get the offset between keys
			offset = key - previousKey;

			// add the position for empty keys
			for (int j = 0; j < offset - 1; j++)
			{
				featureIdListIndex.push_back(keyPosition);
			}

			// update the current position
			keyPosition += numberOfFeatures;
			// add the position for the current key
			featureIdListIndex.push_back(keyPosition);

			// update the previous position
			previousKey = key;
			KeyAndIndex value;
			value.key = key;
			value.index = keyPosition;
			hdmap.map(key, value);
		}

		// fill the key positions for the empty keys
		int maxNumberOfList = this->invert_list_spec_host.totalDimension
				* _maxValuePerDim;
		offset = maxNumberOfList - key - 1;
		for (int j = 0; j < offset; j++)
		{
			featureIdListIndex.push_back(keyPosition);
		}

		inFile.close();
		inFile.clear();
		hdmap.freeze();
		hdmap.show_info("HDMAP");
	}

//		featureIdListIndex.erase(featureIdListIndex.begin());

	double init_time = (double) (clock() - start) / CLOCKS_PER_SEC;
	cout << "Reading Inverted Lists from disk to Host Memory takes: "
			<< init_time << " seconds. " << endl;
	//invert_list_spec_host.maxFeatureID++;
	_maxFeatureID++;		//exclusive feature Id, all smaller than this value

	start = clock();
	host_vector<InvlistEnt> featureIdList_h(featureIdList);
	host_vector<int> featureIdListIndex_h(featureIdListIndex);

	cout << "size of list featureIdList_h: " << featureIdList_h.size() << endl;

	cout << "size of list  featureIdList_h* sizeof(InvlistEnt): "
			<< featureIdList_h.size() * sizeof(InvlistEnt) << endl;

	cout << "size of list featureIdListIndex_h: " << featureIdListIndex_h.size()
			<< endl;

	cout << "host vector allocated" << endl;
	//d_invert_list =new device_vector<device_vector>;
	copyVectorFromHostToDevice(featureIdList_h, d_invert_list);
	//copyVectorFromHostToDevice(featureIdListIndex_h, d_invert_list_idx);

	init_time = (double) (clock() - start) / CLOCKS_PER_SEC;
	cout << "Loading inverted index from CPU to GPU takes: " << init_time
			<< " seconds. " << endl;

}

/**
 * generate random inverted list
 */
void GPUManager::rand_inv_list()
{
	vector<vector<int> > keyword_feature(
			invert_list_spec_host.totalDimension * max_value_per_dimension);
	for (int i = 0; i < invert_list_spec_host.maxFeatureID; i++)
	{
		for (int x = 0; x < invert_list_spec_host.totalDimension; x++)
		{
			int dim_range =
					invert_list_spec_host.indexDimensionEntry[x].maxDomain
							- invert_list_spec_host.indexDimensionEntry[x].minDomain;
			int dim_base =
					invert_list_spec_host.indexDimensionEntry[x].minDomain;
			int dim_value = rand() % dim_range + dim_base;
			int keyword = x * max_value_per_dimension + dim_value;
			keyword_feature[keyword].push_back(i);
		}
	}

	host_vector<InvlistEnt> h_invert_list;
	host_vector<int> h_invert_list_idx;
	int count = 0;
	// construct inverted list
	for (int i = 0;
			i < invert_list_spec_host.totalDimension * max_value_per_dimension;
			i++)
	{
		for (int j = 0; j < keyword_feature[i].size(); j++)
		{
			InvlistEnt new_ent;
			new_ent = keyword_feature[i][j];
			h_invert_list.push_back(new_ent);
		}
		count += keyword_feature[i].size();
		h_invert_list_idx.push_back(count);
	}

	// transfer to gpu
	copyVectorFromHostToDevice(h_invert_list, d_invert_list);
	//copyVectorFromHostToDevice(h_invert_list_idx, d_invert_list_idx);
}

/**
 * read query from file, the query is a list of vectors, i.e. no additional informatino is provided such as weight and bound
 */
void GPUManager::readQueryFromFile(string queryFileName,
		vector<GpuQuery>& _query_set)
{

	if (queryFileName.empty())
	{
		return;
	}

	ifstream query_file(queryFileName.c_str());

	if (!query_file.is_open())
	{
		cout << "query file " << queryFileName << " cannot found" << endl;
		exit(1);
	}

	//host_vector<QueryInfo> h_query_info;
	int num_query = 0;

	while (!query_file.eof())
	{
		string input_line;
		getline(query_file, input_line);

		if (input_line.empty())
		{
			continue;
		}

		stringstream ss(input_line);

		// initialize GpuQuery
		GpuQuery gq(num_query, invert_list_spec_host.totalDimension);

		for (int j = 0; j < invert_list_spec_host.totalDimension; j++)
		{
			gq.dimensionSet[j].dimension = j;
			float dim_value = -1;
			ss >> dim_value;
			gq.keywords[j] = dim_value;
		}

		gq.setDefaultDistType(this->default_disfuncType);
		gq.setDefaultSearchBound(invert_list_spec_host.indexDimensionEntry);
		gq.setDefaultDistanceBound(this->default_upwardDistBound,
				this->default_downwardDistBound);
		_query_set.push_back(gq);
		num_query++;
	}

	cout << "query set size: " << _query_set.size() << endl;
	cout << "readQueryFromFile(): finished loading query file" << endl;
}

void GPUManager::print_query_doucment_mapping()
{
	cout << "query document mapping" << endl;
	host_vector<QueryFeatureEnt> host_mapping = (this->d_query_feature);

	for (int i = 0; i < invert_list_spec_host.numOfQuery; i++)
	{
		for (int j = 0; j < invert_list_spec_host.maxFeatureID; j++)
		{
			QueryFeatureEnt entry = host_mapping[i
					* invert_list_spec_host.maxFeatureID + j];
			cout << entry.count << " ";
		}
		cout << endl;
	}
	cout << "++++++++++++++++++++++++++++++++++++++" << endl;
}

void GPUManager::print_invert_list()
{
//	cout << "invert list" << endl;
//	host_vector<InvlistEnt> host_invert_list = (this->d_invert_list);
//	host_vector<InvlistEnt> host_invert_list_idx = (this->d_invert_list_idx);
//	for (int i = 0; i < host_invert_list_idx.size(); i++)
//	{
//		int dim = i / max_value_per_dimension;
//		int value = i % max_value_per_dimension;
//		cout << "(" << dim << "," << value << "):";
//		int start = i == 0 ? 0 : host_invert_list_idx[i - 1];
//		int end = host_invert_list_idx[i];
//		for (int j = start; j < end; j++)
//			cout << host_invert_list[j] << " ";
//		cout << endl;
//	}
	cout << "++++++++++++++++++++++++++++++++++++++" << endl;
}

void GPUManager::print_result_vec(vector<Result>& result_vec)
{
	cout << "result of output: " << endl;
	for (int i = 0; i < result_vec.size(); i++)
	{
		result_vec[i].print_result_entry();
		cout << endl;
	}
	cout << "++++++++++++++++++++++++++++++++++++++" << endl;
}

//GPU open API, provide basic GPU function via these APIs
template<class KEYWORDMAP, class LASTPOSMAP, class DISTFUNC>
void GPUManager::dev_BidirectionExpansion(KEYWORDMAP keywordMap,
		LASTPOSMAP lastPosMap, DISTFUNC distFunc)
{
	compute_mapping_saving_pos_KernelPerDim_template<<<sumQueryDims,
			THREAD_PER_BLK>>>(raw_pointer_cast(this->d_query_info.data()),
			raw_pointer_cast(this->d_invert_list.data()),
			raw_pointer_cast(this->hdmap.raw_value().data()), //raw_pointer_cast(this->d_invert_list_idx.data()),
			raw_pointer_cast(this->d_query_feature.data()),	//QueryFeatureEnt* query_feature, i.e. count&ACD table
			false,
			max_value_per_dimension,//logical unit, caculated by the number of bits allocated for value
			raw_pointer_cast(this->d_indexDimensionEntry_vec.data()),
			keywordMap, lastPosMap, distFunc);
}

template void GPUManager::dev_BidirectionExpansion<
		DataToIndex_keywordMap_bucketUnit, IndexToData_lastPosMap_bucketUnit,
		Lp_distance>(DataToIndex_keywordMap_bucketUnit keywordMap,
		IndexToData_lastPosMap_bucketUnit lastPosMap, Lp_distance distFunc);

template void GPUManager::dev_BidirectionExpansion<
		DataToIndex_keywordMap_bucket, IndexToData_lastPosMap_bucket_exclusive,
		Lp_distance>(DataToIndex_keywordMap_bucket keywordMap,
		IndexToData_lastPosMap_bucket_exclusive lastPosMap,
		Lp_distance distFunc);

void GPUManager::update_windowQuery_entryAndTable(QueryInfo* host_queryInfo,
		int d_query_info_id)
{

	update_QueryInfo_entry_fromHostToDevice(host_queryInfo,
			this->d_query_info[d_query_info_id]);
	delete h_query_info[d_query_info_id];
	h_query_info[d_query_info_id] = host_queryInfo;

	int wqi_start = d_query_info_id * this->getMaxFeatureID();
	int wqi_end = wqi_start + this->getMaxFeatureID();

	thrust::fill(this->d_query_feature.begin() + wqi_start,
			this->d_query_feature.begin() + wqi_end, QueryFeatureEnt());//clear the Count&ACD table of this window query

}

void GPUManager::update_windowQuery_entryAndTable(GpuQuery& windowQuery)
{

	QueryInfo* hqi = new QueryInfo(windowQuery);

	update_windowQuery_entryAndTable(hqi, windowQuery.queryID);

	//delete hqi;

}

void GPUManager::update_QueryInfo_entry_upperAndLowerBound(int queryId,
		vector<float>& new_upperBoundDist, vector<float>& new_lowerBoundDist)
{

	//update device
	update_QueryInfo_upperAndLowerBound_fromHostToDevice(
			this->d_query_info[queryId], new_upperBoundDist,
			new_lowerBoundDist);
	//update host
	for (int i = 0; i < this->h_query_info[queryId]->numOfDimensionToSearch;
			i++)
	{
		h_query_info[queryId]->upperBoundDist[i] = new_upperBoundDist[i];
		h_query_info[queryId]->lowerBoundDist[i] = new_lowerBoundDist[i];
	}

}

//================================private function

/**
 *
 */
void GPUManager::update_QueryInfo_entry_fromHostToDevice(
		QueryInfo* host_queryInfo, QueryInfo* device_queryInfo)
{

	//copy data

	hipMemcpy(&(device_queryInfo->topK), &(host_queryInfo->topK), sizeof(int),
			hipMemcpyHostToDevice);
	hipMemcpy(&(device_queryInfo->numOfDimensionToSearch),
			&(host_queryInfo->numOfDimensionToSearch), sizeof(int),
			hipMemcpyHostToDevice);
	hipMemcpy(&(device_queryInfo->aggregateFunc),
			&(host_queryInfo->aggregateFunc), sizeof(int),
			hipMemcpyHostToDevice);

	int size = host_queryInfo->numOfDimensionToSearch;
	QueryInfo* d2h_queryInfo = (QueryInfo*) malloc(sizeof(QueryInfo));
	hipMemcpy(d2h_queryInfo, device_queryInfo, sizeof(QueryInfo),
			hipMemcpyDeviceToHost);

	hipMemcpy(d2h_queryInfo->searchDim, host_queryInfo->searchDim,
			sizeof(int) * size, hipMemcpyHostToDevice);
	hipMemcpy(d2h_queryInfo->distanceFunc, host_queryInfo->distanceFunc,
			sizeof(float) * size, hipMemcpyHostToDevice);
	hipMemcpy(d2h_queryInfo->upperBoundSearch,
			host_queryInfo->upperBoundSearch, sizeof(int) * size,
			hipMemcpyHostToDevice);
	hipMemcpy(d2h_queryInfo->lowerBoundSearch,
			host_queryInfo->lowerBoundSearch, sizeof(int) * size,
			hipMemcpyHostToDevice);
	hipMemcpy(d2h_queryInfo->lastPos, host_queryInfo->lastPos,
			sizeof(int2) * size, hipMemcpyHostToDevice);
	hipMemcpy(d2h_queryInfo->keyword, host_queryInfo->keyword,
			sizeof(float) * size, hipMemcpyHostToDevice);
	hipMemcpy(d2h_queryInfo->dimWeight, host_queryInfo->dimWeight,
			sizeof(float) * size, hipMemcpyHostToDevice);
	hipMemcpy(d2h_queryInfo->upperBoundDist, host_queryInfo->upperBoundDist,
			sizeof(float) * size, hipMemcpyHostToDevice);
	hipMemcpy(d2h_queryInfo->lowerBoundDist, host_queryInfo->lowerBoundDist,
			sizeof(float) * size, hipMemcpyHostToDevice);
	free(d2h_queryInfo);

}

void GPUManager::update_QueryInfo_upperAndLowerBound_fromHostToDevice(
		QueryInfo* device_queryInfo, vector<float>& new_upperBoundDist,
		vector<float>& new_lowerBoundDist)
{

	int size = new_upperBoundDist.size();
	QueryInfo* d2h_queryInfo = (QueryInfo*) malloc(sizeof(QueryInfo));
	hipMemcpy(d2h_queryInfo, device_queryInfo, sizeof(QueryInfo),
			hipMemcpyDeviceToHost);

	hipMemcpy(d2h_queryInfo->upperBoundDist, new_upperBoundDist.data(),
			sizeof(float) * size, hipMemcpyHostToDevice);
	hipMemcpy(d2h_queryInfo->lowerBoundDist, new_lowerBoundDist.data(),
			sizeof(float) * size, hipMemcpyHostToDevice);

	free(d2h_queryInfo);
}

void GPUManager::copy_QueryInfo_vec_fromHostToDevice(
		host_vector<QueryInfo*>& hvec, device_vector<QueryInfo*>& _dvec)
{

	for (int i = 0; i < hvec.size(); i++)
	{
		QueryInfo* hostQueryInfo = hvec[i];
		int size = hostQueryInfo->numOfDimensionToSearch;

		// copy class to gpu
		QueryInfo* devQueryInfo;
		hipMalloc((void **) &devQueryInfo, sizeof(QueryInfo));
		hipMemcpy(devQueryInfo, hostQueryInfo, sizeof(QueryInfo),
				hipMemcpyHostToDevice);

		// copy real data to gpu
		int *searchDim;
		float *distanceFunc;
		int *upperBoundSearch;
		int *lowerBoundSearch;
		int2 *lastPos;
		float *keyword;
		float *dimWeight;
		float *upperBoundDist;
		float *lowerBoundDist;
		int *keyword_indexMapping;

		hipMalloc((void **) &searchDim, sizeof(int) * size);
		hipMalloc((void **) &distanceFunc, sizeof(int) * size);
		hipMalloc((void **) &upperBoundSearch, sizeof(int) * size);
		hipMalloc((void **) &lowerBoundSearch, sizeof(int) * size);
		hipMalloc((void **) &lastPos, sizeof(int2) * size);
		hipMalloc((void **) &keyword, sizeof(float) * size);
		hipMalloc((void **) &dimWeight, sizeof(float) * size);
		hipMalloc((void **) &upperBoundDist, sizeof(float) * size);
		hipMalloc((void **) &lowerBoundDist, sizeof(float) * size);
		hipMalloc((void **) &keyword_indexMapping, sizeof(int) * size);

		hipMemcpy(searchDim, hostQueryInfo->searchDim, sizeof(int) * size,
				hipMemcpyHostToDevice);
		hipMemcpy(distanceFunc, hostQueryInfo->distanceFunc,
				sizeof(float) * size, hipMemcpyHostToDevice);
		hipMemcpy(upperBoundSearch, hostQueryInfo->upperBoundSearch,
				sizeof(int) * size, hipMemcpyHostToDevice);
		hipMemcpy(lowerBoundSearch, hostQueryInfo->lowerBoundSearch,
				sizeof(int) * size, hipMemcpyHostToDevice);
		hipMemcpy(lastPos, hostQueryInfo->lastPos, sizeof(int2) * size,
				hipMemcpyHostToDevice);
		hipMemcpy(keyword, hostQueryInfo->keyword, sizeof(float) * size,
				hipMemcpyHostToDevice);
		hipMemcpy(dimWeight, hostQueryInfo->dimWeight, sizeof(float) * size,
				hipMemcpyHostToDevice);
		hipMemcpy(upperBoundDist, hostQueryInfo->upperBoundDist,
				sizeof(float) * size, hipMemcpyHostToDevice);
		hipMemcpy(lowerBoundDist, hostQueryInfo->lowerBoundDist,
				sizeof(float) * size, hipMemcpyHostToDevice);
		hipMemcpy(keyword_indexMapping, hostQueryInfo->keyword_indexMapping,
				sizeof(int) * size, hipMemcpyHostToDevice);

		// copy pointer to gpu
		hipMemcpy(&(devQueryInfo->searchDim), &searchDim, sizeof(int *),
				hipMemcpyHostToDevice);
		hipMemcpy(&(devQueryInfo->distanceFunc), &distanceFunc,
				sizeof(float *), hipMemcpyHostToDevice);
		hipMemcpy(&(devQueryInfo->upperBoundSearch), &upperBoundSearch,
				sizeof(int *), hipMemcpyHostToDevice);
		hipMemcpy(&(devQueryInfo->lowerBoundSearch), &lowerBoundSearch,
				sizeof(int *), hipMemcpyHostToDevice);
		hipMemcpy(&(devQueryInfo->lastPos), &lastPos, sizeof(int2 *),
				hipMemcpyHostToDevice);
		hipMemcpy(&(devQueryInfo->keyword), &keyword, sizeof(float *),
				hipMemcpyHostToDevice);
		hipMemcpy(&(devQueryInfo->dimWeight), &dimWeight, sizeof(float *),
				hipMemcpyHostToDevice);
		hipMemcpy(&(devQueryInfo->upperBoundDist), &upperBoundDist,
				sizeof(float *), hipMemcpyHostToDevice);
		hipMemcpy(&(devQueryInfo->lowerBoundDist), &lowerBoundDist,
				sizeof(float *), hipMemcpyHostToDevice);
		hipMemcpy(&(devQueryInfo->keyword_indexMapping), &keyword_indexMapping,
				sizeof(int *), hipMemcpyHostToDevice);

		_dvec.push_back(devQueryInfo);

	}

}

void GPUManager::free_queryInfo_vec_onDevice(device_vector<QueryInfo*>& _dvec)
{

	for (int i = 0; i < _dvec.size(); i++)
	{
		// copy class reference to cpu
		QueryInfo* devQueryInfo = _dvec[i];

		int *searchDim;
		hipMemcpy(&searchDim, &(devQueryInfo->searchDim), sizeof(int *),
				hipMemcpyDeviceToHost);
		hipFree(searchDim);

		float *distanceFunc;
		hipMemcpy(&distanceFunc, &(devQueryInfo->distanceFunc),
				sizeof(float *), hipMemcpyDeviceToHost);
		hipFree(distanceFunc);

		int *upperBoundSearch;
		hipMemcpy(&upperBoundSearch, &(devQueryInfo->upperBoundSearch),
				sizeof(int *), hipMemcpyDeviceToHost);
		hipFree(upperBoundSearch);

		int *lowerBoundSearch;
		hipMemcpy(&lowerBoundSearch, &(devQueryInfo->lowerBoundSearch),
				sizeof(int *), hipMemcpyDeviceToHost);
		hipFree(lowerBoundSearch);

		int2 *lastPos;
		hipMemcpy(&lastPos, &(devQueryInfo->lastPos), sizeof(int2 *),
				hipMemcpyDeviceToHost);
		hipFree(lastPos);

		float *keyword;
		hipMemcpy(&keyword, &(devQueryInfo->keyword), sizeof(float *),
				hipMemcpyDeviceToHost);
		hipFree(keyword);

		float *dimWeight;
		hipMemcpy(&dimWeight, &(devQueryInfo->dimWeight), sizeof(float *),
				hipMemcpyDeviceToHost);
		hipFree(dimWeight);

		float *upperBoundDist;
		hipMemcpy(&upperBoundDist, &(devQueryInfo->upperBoundDist),
				sizeof(float *), hipMemcpyDeviceToHost);
		hipFree(upperBoundDist);

		float *lowerBoundDist;
		hipMemcpy(&lowerBoundDist, &(devQueryInfo->lowerBoundDist),
				sizeof(float *), hipMemcpyDeviceToHost);
		hipFree(lowerBoundDist);

		hipFree(devQueryInfo);
	}

}

template<class T>
void GPUManager::copyVectorFromHostToDevice(host_vector<T>& hvec,
		device_vector<T> & _dvec)
{
	//_dvec = new device_vector<T>;
	//_dvec->reserve(hvec.size());

	//for ( int i = 0; i < hvec.size(); i++ )
	//{
	//	_dvec[i] = hvec[i];
	//_dvec->push_back(hvec[i]);
	//}

	_dvec = hvec;

}

void GPUManager::check_query_parameter()
{

	if (!this->query_initialized)
	{
		cout << "query has not been initialized" << endl;
		exit(1);
	}

	if (THREAD_PER_BLK < invert_list_spec_host.totalDimension)
	{
		cout << "thread per block must be larger than max dimension of x"
				<< endl;
		exit(1);
	}
	if (this->invert_list_spec_host.numOfQuery > 65535)
	{
		cout
				<< "number of query should not exceed 65535: need to modify the code"
				<< endl;
		exit(1);
	}
	if (THREAD_PER_BLK > 1024)
	{
		cout << "THREAD_PER_BLK should not exceed 1024: need to modify the code"
				<< endl;
		exit(1);
	}

}

void GPUManager::printPrunStatistics()
{

	int sci = 0;
	for (int i = 0; i < this->invert_list_spec_host.totalDimension; i++)
	{
		int ci = thrust::count_if(d_query_feature.begin(),
				d_query_feature.end(), gCount(i));
		//cout<<"The item with count["<<i<<"] is:"<<ci<<endl;
		sci += ci;
	}
	cout << "the total item touched is:" << sci << endl;

	QueryFeatureEnt init;
	init.count = 0;
	init.ACD = 0.f;
	QueryFeatureEnt sum = thrust::reduce(d_query_feature.begin(),
			d_query_feature.end(), init, countFeature());
	cout << "the average sum of count table (touched) per query is:"
			<< sum.count / d_query_info.size() << endl;
	cout << "the total number of items in the idnex is:" << d_invert_list.size()
			<< endl;
	cout << "prune rate (1-touched/total) ="
			<< 1
					- ((double) sum.count / d_query_info.size())
							/ d_invert_list.size() << endl;
}

void GPUManager::runTest()
{
	int blk_num = 2;
	int topKValue = 7;

	host_vector<Result> h_result(THREAD_PER_BLK * blk_num);
	host_vector<Result> h_result_temp(THREAD_PER_BLK * blk_num);
	host_vector<int> h_query_result_count(blk_num);
	h_query_result_count[0] = THREAD_PER_BLK;
	h_query_result_count[1] = 2 * THREAD_PER_BLK;
	host_vector<int> h_valid_query;
	h_valid_query.resize(blk_num, 1);

	device_vector<Result> d_result(
			h_query_result_count[h_query_result_count.size() - 1]);
	device_vector<Result> d_result_temp(
			h_query_result_count[h_query_result_count.size() - 1]);
	device_vector<int> d_query_result_count(blk_num);
	device_vector<int> d_valid_query(blk_num);

	for (int i = 0; i < blk_num * THREAD_PER_BLK; i++)
	{

		h_result[i].count = i;	//rand()%(blk_num*THREAD_PER_BLK);
		h_result[i].lb = rand() % (blk_num * THREAD_PER_BLK);
		h_result[i].ub = rand() % (blk_num * THREAD_PER_BLK);

	}

	h_result[55].lb = -1;
	h_result[66].ub = -3;

	h_result[200].lb = -8;
	h_result[180].ub = -9;

	d_result = h_result;
	d_query_result_count = h_query_result_count;
	d_valid_query = h_valid_query;

	terminateCheck_kSelection_KernelPerQuery<<<blk_num, THREAD_PER_BLK / 5>>>(
			raw_pointer_cast(d_result.data()),
			raw_pointer_cast(d_result_temp.data()),
			raw_pointer_cast(d_query_result_count.data()),
			raw_pointer_cast(d_valid_query.data()), topKValue);

	h_result = d_result;

	for (int j = 0; j < blk_num; j++)
	{
		cout << "BLK =" << j << endl;
		for (int i = j * THREAD_PER_BLK; i < j * THREAD_PER_BLK + topKValue + 3;
				i++)
		{
			cout << " d_result_ub_sorted[" << h_result[i].count << "].ub ="
					<< h_result[i].ub;
		}
		cout << endl;

		for (int i = j * THREAD_PER_BLK; i < j * THREAD_PER_BLK + topKValue + 3;
				i++)
		{
			cout << " d_result_ub_sorted[" << h_result[i].count << "].lb ="
					<< h_result[i].lb;
		}
		cout << endl;

	}
}

//junk code

//==========================junk code  2014.06.10

/*

 //================================== v1 of bi_direction_query_KernelPerQuery
 *
 * each query take one block (kernel), therefore this function is query oriented parallel computing, this is an old version

 bool GPUManager::bi_direction_query_KernelPerQuery ( int threshold, int topKValue, vector<Result>& _result )
 {

 check_query_parameter();

 hipProfilerStart();
 hipEvent_t start, stop;
 hipEvent_t gs,ge;
 float elapsedTime;
 float elapseTimeForCount;
 hipEventCreate(&start);
 hipEventCreate(&stop);
 hipEventCreate(&gs);
 hipEventCreate(&ge);


 int terminate_sum;

 hipEventRecord(gs, 0);
 cout << "GPUManager::bi_direction_query(): 0 -- compute_mapping_saving_pos_KernelPerQuery()" << endl;
 hipEventRecord(start, 0);
 compute_mapping_saving_pos_KernelPerQuery<<<this->invert_list_spec_host.numOfQuery, THREAD_PER_BLK, max_number_of_dimensions*sizeof(int2) >>> (
 raw_pointer_cast( this->d_query_info.data() ),
 raw_pointer_cast( this->d_invert_list.data() ),
 raw_pointer_cast( this->d_invert_list_idx.data() ),
 raw_pointer_cast( this->d_query_feature.data() ),//QueryFeatureEnt* query_feature, count&ACD table
 false,
 max_value_per_dimension,
 raw_pointer_cast( this->d_vec_indexDimensionEntry.data() )
 );
 //hipDeviceSynchronize();
 hipEventRecord(stop, 0);
 hipEventSynchronize(stop);
 hipEventElapsedTime(&elapsedTime, start, stop);
 exec_time[0] += elapsedTime;

 device_vector<int> threshold_count(invert_list_spec_host.numOfQuery * THREAD_PER_BLK, 0);	// prefix count for each thread
 device_vector<int> query_result_count(invert_list_spec_host.numOfQuery,0);				// prefix count for each block (i.e. each query)

 cout << "GPUManager::bi_direction_query(): 1 -- prefix_count_KernelPerQuery()" << endl;
 elapseTimeForCount = 0;
 hipEventRecord(start, 0);
 prefix_count_KernelPerQuery<<<this->invert_list_spec_host.numOfQuery,THREAD_PER_BLK>>>(
 raw_pointer_cast(this->d_query_feature.data()),
 raw_pointer_cast(threshold_count.data()),
 raw_pointer_cast(query_result_count.data()),
 threshold
 );
 hipEventRecord(stop, 0);
 hipEventSynchronize(stop);
 hipEventElapsedTime(&elapsedTime, start, stop);
 elapseTimeForCount += elapsedTime;
 exec_time[7] += elapsedTime;

 hipEventRecord(start, 0);
 thrust::inclusive_scan(threshold_count.begin(), threshold_count.end(), threshold_count.begin()); // per thread inclusive scan
 hipEventRecord(stop, 0);
 hipEventSynchronize(stop);
 hipEventElapsedTime(&elapsedTime, start, stop);
 exec_time[8] += elapsedTime;
 elapseTimeForCount += elapsedTime;

 hipEventRecord(start, 0);
 thrust::inclusive_scan(query_result_count.begin(), query_result_count.end(), query_result_count.begin()); // per block inclusive scan
 hipEventRecord(stop, 0);
 hipEventSynchronize(stop);
 hipEventElapsedTime(&elapsedTime, start, stop);
 exec_time[9] += elapsedTime;
 elapseTimeForCount += elapsedTime;
 exec_time[1] += elapseTimeForCount;


 cout << "GPUManager::bi_direction_query(): 2 -- output_result_bidrection_search_KernelPerQuery()" << endl;
 device_vector<Result> d_result_ub_sorted(threshold_count[threshold_count.size()-1]);


 hipEventRecord(start, 0);
 output_result_bidrection_search_KernelPerQuery<<< this->invert_list_spec_host.numOfQuery,THREAD_PER_BLK, 3*max_number_of_dimensions*sizeof(float) >>>(
 raw_pointer_cast(this->d_query_feature.data()),
 raw_pointer_cast(this->d_query_info.data()),
 raw_pointer_cast(threshold_count.data()),
 raw_pointer_cast(d_result_ub_sorted.data()),//record the lower bound and upper bound
 threshold,
 raw_pointer_cast( this->d_vec_indexDimensionEntry.data() )
 );
 hipEventRecord(stop, 0);
 hipEventSynchronize(stop);
 hipEventElapsedTime(&elapsedTime, start, stop);
 exec_time[2] += elapsedTime;

 device_vector<Result> d_result_lb_sorted(d_result_ub_sorted.begin(),d_result_ub_sorted.end());


 cout << "GPUManager::bi_direction_query(): 3 -- sort d_result_lb & ub " << endl;
 hipEventRecord(start, 0);
 thrust::sort( d_result_ub_sorted.begin(), d_result_ub_sorted.end(), Ubcomapre() );
 //Result_kSelectionSort()


 thrust::sort( d_result_lb_sorted.begin(), d_result_lb_sorted.end(), Lbcompare() );
 hipEventRecord(stop, 0);
 hipEventSynchronize(stop);
 hipEventElapsedTime(&elapsedTime, start, stop);
 exec_time[3] += elapsedTime;


 cout << "GPUManager::bi_direction_query(): 4 -- terminate_check_KernelPerQuery()" << endl;
 hipEventRecord(start, 0);
 terminateCheck_afterSort_KernelPerQuery<<<this->invert_list_spec_host.numOfQuery,THREAD_PER_BLK>>>(
 raw_pointer_cast(d_result_ub_sorted.data()),
 raw_pointer_cast(d_result_lb_sorted.data()),
 raw_pointer_cast(query_result_count.data()),
 raw_pointer_cast(d_valid_query.data()),
 topKValue,
 this->num_of_rounds);
 terminate_sum = thrust::reduce(d_valid_query.begin(),d_valid_query.end());
 hipEventRecord(stop, 0);
 hipEventSynchronize(stop);
 hipEventElapsedTime(&elapsedTime, start, stop);
 exec_time[4] += elapsedTime;

 cout << "GPUManager::bi_direction_query(): 5 -- extract_topK_KernelPerQuery()" << endl;
 device_vector<Result> d_result( this->invert_list_spec_host.numOfQuery * topKValue );
 hipEventRecord(start, 0);
 extract_topK_KernelPerQuery<<< this->invert_list_spec_host.numOfQuery, THREAD_PER_BLK >>>(
 raw_pointer_cast(d_result_ub_sorted.data()),
 raw_pointer_cast(query_result_count.data()),
 raw_pointer_cast(d_result.data()),
 topKValue);
 hipEventRecord(stop, 0);
 hipEventSynchronize(stop);
 hipEventElapsedTime(&elapsedTime, start, stop);
 exec_time[5] += elapsedTime;


 hipEventRecord(ge, 0);
 hipEventSynchronize(ge);
 hipEventElapsedTime(&elapsedTime, gs, ge);
 exec_time[6] += elapsedTime;

 hipEventDestroy(start);
 hipEventDestroy(stop);
 hipEventDestroy(gs);
 hipEventDestroy(ge);

 hipProfilerStop();



 //	for ( int index = 0 ; index < d_result_ub_sorted.size(); index ++ )
 //	{
 //		Result tempResult = d_result_ub_sorted[index];
 //		tempResult.print_result_entry();
 //		printf("\n");
 //	}



 num_of_rounds++;

 _result.resize(d_result.size());
 HANDLE_ERROR( hipMemcpy( &_result[0], raw_pointer_cast(d_result.data()), d_result.size()*sizeof(Result), hipMemcpyDeviceToHost) );

 // no more query should be made to this batch
 if (terminate_sum == 0) {
 this->query_initialized = false;
 }


 return terminate_sum == 0;
 }
 */

/**
 *
 *
 bool GPUManager::bi_direction_query_KernelPerDim_V1 (int threshold, int topKValue, vector<Result>& _result )
 {

 check_query_parameter();

 hipProfilerStart();
 hipEvent_t start, stop;
 hipEvent_t gs, ge;
 float elapsedTime;
 float elapseTimeForCount;
 hipEventCreate(&start);
 hipEventCreate(&stop);
 hipEventCreate(&gs);
 hipEventCreate(&ge);

 cout	<< "GPUManager::bi_direction_query(): 0 -- compute_mapping_saving_pos_KernelPerDim()"
 << endl;
 hipEventRecord(gs, 0);
 hipEventRecord(start, 0);
 compute_mapping_saving_pos_KernelPerDim<<<sumQueryDims, THREAD_PER_BLK>>>(
 raw_pointer_cast(this->d_query_info.data()),
 raw_pointer_cast(this->d_invert_list.data()),
 raw_pointer_cast(this->d_invert_list_idx.data()),
 raw_pointer_cast(this->d_query_feature.data()),	//QueryFeatureEnt* query_feature, count&ACD table
 false, max_value_per_dimension,
 raw_pointer_cast(this->minDomainForAllDimension.data()),
 raw_pointer_cast(this->maxDomainForAllDimension.data()));
 hipEventRecord(stop, 0);
 hipEventSynchronize(stop);
 hipEventElapsedTime(&elapsedTime, start, stop);
 exec_time[0] += elapsedTime;

 cout	<< "GPUManager::bi_direction_query(): 1 -- prefix_count_KernelPerQuery()"
 << endl;
 device_vector<int> threshold_count(	invert_list_spec_host.numOfQuery * THREAD_PER_BLK, 0);// prefix count for each thread
 device_vector<int> query_result_count(invert_list_spec_host.numOfQuery, 0);	// prefix count for each block (i.e. each query)
 elapseTimeForCount=0;

 hipEventRecord(start, 0);
 depressed_prefix_count_KernelPerQuery_V1<<<this->invert_list_spec_host.numOfQuery,
 THREAD_PER_BLK>>>(raw_pointer_cast(this->d_query_feature.data()),
 raw_pointer_cast(threshold_count.data()),
 raw_pointer_cast(query_result_count.data()), threshold,
 raw_pointer_cast(this->minDomainForAllDimension.data()),
 raw_pointer_cast(this->maxDomainForAllDimension.data()));
 hipEventRecord(stop, 0);
 hipEventSynchronize(stop);
 hipEventElapsedTime(&elapsedTime, start, stop);
 elapseTimeForCount+=elapsedTime;
 exec_time[7] += elapsedTime;

 hipEventRecord(start, 0);
 thrust::inclusive_scan(threshold_count.begin(), threshold_count.end(),
 threshold_count.begin()); // per thread inclusive scan
 hipEventRecord(stop, 0);
 hipEventSynchronize(stop);
 hipEventElapsedTime(&elapsedTime, start, stop);
 elapseTimeForCount+=elapsedTime;
 exec_time[8] += elapsedTime;
 hipEventRecord(start, 0);
 thrust::inclusive_scan(query_result_count.begin(), query_result_count.end(),
 query_result_count.begin()); // per block inclusive scan
 hipEventRecord(stop, 0);
 hipEventSynchronize(stop);
 hipEventElapsedTime(&elapsedTime, start, stop);
 elapseTimeForCount+=elapsedTime;
 exec_time[9] += elapsedTime;
 exec_time[1] += elapseTimeForCount;

 cout	<< "GPUManager::bi_direction_query(): 2 -- output_result_bidrection_search_KernelPerQuery()"
 << endl;
 device_vector<Result> d_result_ub_sorted(
 threshold_count[threshold_count.size() - 1]);

 hipEventRecord(start, 0);
 output_result_bidrection_search_KernelPerQuery_V1<<<
 this->invert_list_spec_host.numOfQuery, THREAD_PER_BLK,
 3 * max_number_of_dimensions*sizeof(float)>>>(
 raw_pointer_cast(this->d_query_feature.data()),
 raw_pointer_cast(this->d_query_info.data()),
 raw_pointer_cast(threshold_count.data()),
 raw_pointer_cast(d_result_ub_sorted.data()), //record the lower bound and upper bound
 threshold, raw_pointer_cast(this->minDomainForAllDimension.data()),
 raw_pointer_cast(this->maxDomainForAllDimension.data()));
 hipEventRecord(stop, 0);
 hipEventSynchronize(stop);
 hipEventElapsedTime(&elapsedTime, start, stop);
 exec_time[2] += elapsedTime;

 cout << "GPUManager::bi_direction_query(): 3 -- sort d_result_lb & ub"
 << endl;
 device_vector<Result> d_result_lb_sorted(d_result_ub_sorted.begin(),
 d_result_ub_sorted.end());
 hipEventRecord(start, 0);
 thrust::sort(d_result_ub_sorted.begin(), d_result_ub_sorted.end(),
 Ubcomapre());
 thrust::sort(d_result_lb_sorted.begin(), d_result_lb_sorted.end(),
 Lbcompare());
 hipEventRecord(stop, 0);
 hipEventSynchronize(stop);
 hipEventElapsedTime(&elapsedTime, start, stop);
 exec_time[3] += elapsedTime;

 cout
 << "GPUManager::bi_direction_query(): 4 -- terminate_check_KernelPerQuery()"
 << endl;
 hipEventRecord(start, 0);
 terminate_check_KernelPerQuery_V1<<<this->invert_list_spec_host.numOfQuery,
 THREAD_PER_BLK>>>(raw_pointer_cast(d_result_ub_sorted.data()),
 raw_pointer_cast(d_result_lb_sorted.data()),
 raw_pointer_cast(query_result_count.data()),
 raw_pointer_cast(d_valid_query.data()), topKValue,
 this->num_of_rounds);
 int terminate_sum = thrust::reduce(d_valid_query.begin(),
 d_valid_query.end());
 hipEventRecord(stop, 0);
 hipEventSynchronize(stop);
 hipEventElapsedTime(&elapsedTime, start, stop);
 exec_time[4] += elapsedTime;

 cout
 << "GPUManager::bi_direction_query(): 4 -- extract_topK_KernelPerQuery()"
 << endl;
 device_vector<Result> d_result(
 this->invert_list_spec_host.numOfQuery * topKValue);
 hipEventRecord(start, 0);
 extract_topK_KernelPerQuery<<<this->invert_list_spec_host.numOfQuery,
 THREAD_PER_BLK>>>(raw_pointer_cast(d_result_ub_sorted.data()),
 raw_pointer_cast(query_result_count.data()),
 raw_pointer_cast(d_result.data()), topKValue);
 hipEventRecord(stop, 0);
 hipEventSynchronize(stop);
 hipEventElapsedTime(&elapsedTime, start, stop);
 exec_time[5] += elapsedTime;

 hipEventRecord(ge, 0);
 hipEventSynchronize(ge);
 hipEventElapsedTime(&elapsedTime, gs, ge);
 exec_time[6] += elapsedTime;

 hipEventDestroy(start);
 hipEventDestroy(stop);
 hipProfilerStop();

 num_of_rounds++;

 _result.resize(d_result.size());
 HANDLE_ERROR(
 hipMemcpy(&_result[0], raw_pointer_cast(d_result.data()),
 d_result.size() * sizeof(Result), hipMemcpyDeviceToHost));

 // no more query should be made to this batch
 if (terminate_sum == 0) {
 this->query_initialized = false;
 }

 return terminate_sum == 0;
 }
 *
 *
 *
 */

/**
 *
 *

 //================================================= V2 of bi_direction_query_KernelPerQuery
 **
 * each query take one block (kernel), therefore this function is query oriented parallel computing, this is an old version
 *
 bool GPUManager::bi_direction_query_KernelPerQuery_V2 ( int threshold, int topKValue, vector<Result>& _result )
 {

 check_query_parameter();

 hipProfilerStart();
 hipEvent_t start, stop;
 hipEvent_t gs,ge;
 float elapsedTime;
 float elapseTimeForCount;
 hipEventCreate(&start);
 hipEventCreate(&stop);
 hipEventCreate(&gs);
 hipEventCreate(&ge);


 int terminate_sum;

 hipEventRecord(gs, 0);
 cout << "GPUManager::bi_direction_query(): 0 -- compute_mapping_saving_pos_KernelPerQuery()" << endl;
 hipEventRecord(start, 0);
 compute_mapping_saving_pos_KernelPerQuery<<<this->invert_list_spec_host.numOfQuery, THREAD_PER_BLK, max_number_of_dimensions*sizeof(int2) >>> (
 raw_pointer_cast( this->d_query_info.data() ),
 raw_pointer_cast( this->d_invert_list.data() ),
 raw_pointer_cast( this->d_invert_list_idx.data() ),
 raw_pointer_cast( this->d_query_feature.data() ),//QueryFeatureEnt* query_feature, count&ACD table
 false,
 max_value_per_dimension,
 raw_pointer_cast( this->minDomainForAllDimension.data() ),
 raw_pointer_cast( this->maxDomainForAllDimension.data() ) );
 //hipDeviceSynchronize();
 hipEventRecord(stop, 0);
 hipEventSynchronize(stop);
 hipEventElapsedTime(&elapsedTime, start, stop);
 exec_time[0] += elapsedTime;



 device_vector<int> threshold_count(invert_list_spec_host.numOfQuery * THREAD_PER_BLK, 0);	// prefix count for each thread
 device_vector<int> query_result_count(invert_list_spec_host.numOfQuery,0);				// prefix count for each block (i.e. each query)

 cout << "GPUManager::bi_direction_query(): 1 -- prefix_count_KernelPerQuery()" << endl;
 elapseTimeForCount = 0;
 hipEventRecord(start, 0);
 prefix_count_KernelPerQuery<<<this->invert_list_spec_host.numOfQuery,THREAD_PER_BLK,THREAD_PER_BLK*sizeof(int)>>>(
 raw_pointer_cast(this->d_query_feature.data()),
 raw_pointer_cast(threshold_count.data()),
 raw_pointer_cast(query_result_count.data()),
 threshold,
 raw_pointer_cast( this->minDomainForAllDimension.data() ),
 raw_pointer_cast( this->maxDomainForAllDimension.data() ) );
 hipEventRecord(stop, 0);
 hipEventSynchronize(stop);
 hipEventElapsedTime(&elapsedTime, start, stop);
 elapseTimeForCount += elapsedTime;
 exec_time[7] += elapsedTime;

 hipEventRecord(start, 0);
 thrust::inclusive_scan(threshold_count.begin(), threshold_count.end(), threshold_count.begin()); // per thread inclusive scan
 hipEventRecord(stop, 0);
 hipEventSynchronize(stop);
 hipEventElapsedTime(&elapsedTime, start, stop);
 exec_time[8] += elapsedTime;
 elapseTimeForCount += elapsedTime;

 hipEventRecord(start, 0);
 thrust::inclusive_scan(query_result_count.begin(), query_result_count.end(), query_result_count.begin()); // per block inclusive scan
 //int query_result_count_idx = THREAD_PER_BLK -1;
 //for(int i=0;i<query_result_count.size();i++){
 //cout<<"previous query_result_count "<<i<<" "<<query_result_count[i]<<endl;
 //	query_result_count[i] = threshold_count[query_result_count_idx];
 //	query_result_count_idx+=THREAD_PER_BLK;
 //cout<<"after query_result_count "<<i<<" "<<query_result_count[i]<<endl;
 //}

 hipEventRecord(stop, 0);
 hipEventSynchronize(stop);
 hipEventElapsedTime(&elapsedTime, start, stop);
 exec_time[9] += elapsedTime;
 elapseTimeForCount += elapsedTime;
 exec_time[1] += elapseTimeForCount;


 cout << "GPUManager::bi_direction_query(): 2 -- output_result_bidrection_search_KernelPerQuery()" << endl;
 device_vector<Result> d_result_ub_sorted(threshold_count[threshold_count.size()-1]);
 hipEventRecord(start, 0);
 //note: v1 version is just OK
 output_result_bidrection_search_KernelPerQuery_V1<<< this->invert_list_spec_host.numOfQuery,THREAD_PER_BLK, 3*max_number_of_dimensions*sizeof(float) >>>(
 raw_pointer_cast(this->d_query_feature.data()),
 raw_pointer_cast(this->d_query_info.data()),
 raw_pointer_cast(threshold_count.data()),
 //raw_pointer_cast(query_result_count.data()),
 raw_pointer_cast(d_result_ub_sorted.data()),//record the lower bound and upper bound
 threshold,
 raw_pointer_cast( this->minDomainForAllDimension.data() ),
 raw_pointer_cast( this->maxDomainForAllDimension.data() ) );
 hipEventRecord(stop, 0);
 hipEventSynchronize(stop);
 hipEventElapsedTime(&elapsedTime, start, stop);
 exec_time[2] += elapsedTime;

 device_vector<Result> d_result_lb_sorted(d_result_ub_sorted.begin(),d_result_ub_sorted.end());


 cout << "GPUManager::bi_direction_query(): 3 -- sort d_result_lb & ub " << endl;
 hipEventRecord(start, 0);
 //for(int i=0;i<this->invert_list_spec_host.numOfQuery;i++){
 ///int block_start =(i == 0 ? 0 : query_result_count[i - 1]);
 //int block_end = query_result_count[i];
 //thrust::sort( d_result_ub_sorted.begin()+ block_start, d_result_ub_sorted.begin() +block_end, Ubcomapre() );
 //thrust::sort( d_result_lb_sorted.begin()+block_start, d_result_lb_sorted.begin() +block_end, Lbcompare() );
 //}

 thrust::sort( d_result_ub_sorted.begin(), d_result_ub_sorted.end() , Ubcomapre() );
 thrust::sort( d_result_lb_sorted.begin(), d_result_lb_sorted.end() , Lbcompare() );
 hipEventRecord(stop, 0);
 hipEventSynchronize(stop);
 hipEventElapsedTime(&elapsedTime, start, stop);
 exec_time[3] += elapsedTime;


 cout << "GPUManager::bi_direction_query(): 4 -- terminate_check_KernelPerQuery()" << endl;
 hipEventRecord(start, 0);
 terminate_check_KernelPerQuery_V1<<<this->invert_list_spec_host.numOfQuery,THREAD_PER_BLK>>>(
 raw_pointer_cast(d_result_ub_sorted.data()),
 raw_pointer_cast(d_result_lb_sorted.data()),
 raw_pointer_cast(query_result_count.data()),
 raw_pointer_cast(d_valid_query.data()),
 topKValue,
 this->num_of_rounds);
 terminate_sum = thrust::reduce(d_valid_query.begin(),d_valid_query.end());
 hipEventRecord(stop, 0);
 hipEventSynchronize(stop);
 hipEventElapsedTime(&elapsedTime, start, stop);
 exec_time[4] += elapsedTime;

 cout << "GPUManager::bi_direction_query(): 5 -- extract_topK_KernelPerQuery()" << endl;
 device_vector<Result> d_result( this->invert_list_spec_host.numOfQuery * topKValue );
 hipEventRecord(start, 0);
 extract_topK_KernelPerQuery<<< this->invert_list_spec_host.numOfQuery, THREAD_PER_BLK >>>(
 raw_pointer_cast(d_result_ub_sorted.data()),
 raw_pointer_cast(query_result_count.data()),
 raw_pointer_cast(d_result.data()),
 topKValue);
 hipEventRecord(stop, 0);
 hipEventSynchronize(stop);
 hipEventElapsedTime(&elapsedTime, start, stop);
 exec_time[5] += elapsedTime;


 hipEventRecord(ge, 0);
 hipEventSynchronize(ge);
 hipEventElapsedTime(&elapsedTime, gs, ge);
 exec_time[6] += elapsedTime;

 hipEventDestroy(start);
 hipEventDestroy(stop);
 hipEventDestroy(gs);
 hipEventDestroy(ge);

 hipProfilerStop();

 num_of_rounds++;

 _result.resize(d_result.size());
 HANDLE_ERROR( hipMemcpy( &_result[0], raw_pointer_cast(d_result.data()), d_result.size()*sizeof(Result), hipMemcpyDeviceToHost) );

 // no more query should be made to this batch
 if (terminate_sum == 0) {
 this->query_initialized = false;
 }


 return terminate_sum == 0;
 }

 */
